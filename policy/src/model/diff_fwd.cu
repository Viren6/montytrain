
#include <hip/hip_runtime.h>
__forceinline__ __device__ void addf4(float4* dst, const float4 src)
{
    dst->x += src.x;
    dst->y += src.y;
    dst->z += src.z;
    dst->w += src.w;
}

__forceinline__ __device__ void subf4(float4* dst, const float4 src)
{
    dst->x -= src.x;
    dst->y -= src.y;
    dst->z -= src.z;
    dst->w -= src.w;
}

__forceinline__ __device__ float op(const float x)
{
    const float clamp = max(min(x, 1.0F), 0.0F);
    return clamp * clamp;
}

extern "C" __global__ void kernel(
    const int batch_size,
    const int hl_size,
    const float* weights,
    const float* hl,
    const int* moves,
    float* output
) {
    const int loc_in_batch = blockIdx.z;
    const int loc_in_moves = blockIdx.y;
    const int loc_in_neurons = blockIdx.x * blockDim.x + threadIdx.x;

    if (4 * loc_in_neurons >= hl_size || loc_in_batch >= batch_size || loc_in_moves >= 64)
    {
        return;
    }

    const int locmb = 64 * loc_in_batch + loc_in_moves;
    const int4 move = reinterpret_cast<const int4*>(moves)[locmb];

    float4 val = make_float4(0.0F, 0.0F, 0.0F, 0.0F);

    if (move.x != -1) {
        val = reinterpret_cast<const float4*>(hl + hl_size * loc_in_batch)[loc_in_neurons];
        subf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.x)[loc_in_neurons]);

        if (move.y != -1)
        {
            subf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.y)[loc_in_neurons]);
        }

        addf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.z)[loc_in_neurons]);

        if (move.w != -1)
        {
            addf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.w)[loc_in_neurons]);
        }

        val.x = op(val.x);
        val.y = op(val.y);
        val.z = op(val.z);
        val.w = op(val.w);
    }

    reinterpret_cast<float4*>(output + hl_size * locmb)[loc_in_neurons] = val;
}