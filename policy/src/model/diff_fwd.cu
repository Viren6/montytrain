
#include <hip/hip_runtime.h>
__forceinline__ __device__ void addf4(float4* dst, const float4 src)
{
    dst->x += src.x;
    dst->y += src.y;
    dst->z += src.z;
    dst->w += src.w;
}

__forceinline__ __device__ void subf4(float4* dst, const float4 src)
{
    dst->x -= src.x;
    dst->y -= src.y;
    dst->z -= src.z;
    dst->w -= src.w;
}

extern "C" __global__ void kernel(
    const int batch_size,
    const int hl_size,
    const float* weights,
    const float* hl,
    const int* moves,
    float* output
) {
    const int loc_in_batch = blockIdx.z;
    const int loc_in_moves = blockIdx.y;
    const int loc_in_neurons = blockIdx.x * blockDim.x + threadIdx.x;

    if (4 * loc_in_neurons >= hl_size || loc_in_batch >= batch_size || loc_in_moves >= 64)
    {
        return;
    }

    const int4 move = reinterpret_cast<const int4*>(moves)[64 * loc_in_batch + loc_in_moves];

    float4 val = make_float4(0.0F, 0.0F, 0.0F, 0.0F);

    if (move.x + move.y + move.z + move.w != -4) {
        val = reinterpret_cast<const float4*>(hl)[loc_in_neurons];

        if (move.x != -1)
        {
            subf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.x)[loc_in_neurons]);
        }

        if (move.y != -1)
        {
            subf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.y)[loc_in_neurons]);
        }

        if (move.z != -1)
        {
            addf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.z)[loc_in_neurons]);
        }

        if (move.w != -1)
        {
            addf4(&val, reinterpret_cast<const float4*>(weights + hl_size * move.w)[loc_in_neurons]);
        }
    }

    float4* this_output = reinterpret_cast<float4*>(output + 64 * hl_size * loc_in_batch + hl_size * loc_in_moves);
    this_output[loc_in_neurons] = val;
}